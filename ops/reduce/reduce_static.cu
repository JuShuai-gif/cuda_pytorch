#include "hip/hip_runtime.h"
#include "common.h"

#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_runtime_wrapper.h>
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

const int NUM_REPEATS = 100;
const int N = 100000000;
const int M = sizeof(float) * N;
const int BLOCK_SIZE = 128;
const int GRID_SIZE = 10240;

void timing(const float *d_x);

int main(void) {
    float *h_x = (float *)malloc(M);
    for (int n = 0; n < N; ++n) {
        h_x[n] = 1.23;
    }
    float *d_x;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

    timing(d_x);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ reduce_cp(const float *d_x, float *d_y, const int N) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    extern __shared__ float s_y[];

    float y{0.0f};

    const int stride = blockDim.x * gridDim.x;
    for (int n = bid * blockDim.x + tid; n < N; n += stride) {
        y += d_x[n];
    }

    s_y[tid] = y;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1) {
        if (tid < offset) {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    y = s_y[tid];

    thread_block_tile<32> g = tiled_partition<32>(this_thread_block());
    for (int i = g.size() >> 1; i > 0; i >>= 1) {
        y += g.shfl_down(y, i);
    }

    if (tid == 0) {
        d_y[bid] = y;
    }
}

__device__ float static_y[GRID_SIZE];

float reduce(const float *d_x) {
    float *d_y;
    CHECK(hipGetSymbolAddress((void **)&d_y, static_y));

    const int smem = sizeof(float) * BLOCK_SIZE;

    reduce_cp<<<GRID_SIZE, BLOCK_SIZE, smem>>>(d_x, d_y, N);
    reduce_cp<<<1, 1024, sizeof(float) * 1024>>>(d_y, d_y, GRID_SIZE);

    float h_y[1] = {0};
    CHECK(hipMemcpy(h_y, d_y, sizeof(float), hipMemcpyDeviceToHost));

    return h_y[0];
}

void timing(const float *d_x) {
    float sum = 0;

    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat) {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(d_x);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("sum = %f.\n", sum);
}
